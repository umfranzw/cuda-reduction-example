/* Some simple convenience functions to make timing CUDA events a little less verbose.
  */

#include "timer.h"
#include "utils.h"

Timer create_timer()
{
    Timer timer;
    hipEventCreate(&(timer.start));
    hipEventCreate(&(timer.stop));

    return timer;
}

void destroy_timer(Timer *timer)
{
    hipEventDestroy(timer->start);
    hipEventDestroy(timer->stop);
}

void start_timer(Timer *timer)
{
    hipEventRecord(timer->start);
}

void stop_timer(Timer *timer)
{
    hipEventRecord(timer->stop);
}

// Returns the elapsed time between the given events, in milliseconds
float get_time(Timer *timer)
{
    hipError_t status;
    float millisec = 0;
    status = hipEventElapsedTime(&millisec, timer->start, timer->stop);
    check_error(status, "Error calculating time between CUDA events.");
    
    return millisec;
}
